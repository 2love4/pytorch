#include "hip/hip_runtime.h"
#include <limits>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Math.cuh>

namespace at { namespace native {

void acos_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_C10_COMPLEX_TYPES_AND1(ScalarType::Half, iter.dtype(), "acos_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::acos(a);
    });
  });
}

void asin_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_C10_COMPLEX_TYPES_AND1(ScalarType::Half, iter.dtype(), "asin_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::asin(a);
    });
  });
}

void sin_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_C10_COMPLEX_TYPES_AND1(ScalarType::Half, iter.dtype(), "sin_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::sin(a);
    });
  });
}

void cos_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_C10_COMPLEX_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.dtype(), "cos_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::cos(a);
    });
  });
}

void sinh_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_C10_COMPLEX_TYPES_AND1(ScalarType::Half, iter.dtype(), "sinh_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::sinh(a);
    });
  });
}

void cosh_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_C10_COMPLEX_TYPES_AND1(ScalarType::Half, iter.dtype(), "cosh_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::cosh(a);
    });
  });
}

void tanh_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND(ScalarType::Half, iter.dtype(), "tanh_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::tanh(a);
    });
  });
}

REGISTER_DISPATCH(acos_stub, &acos_kernel_cuda);
REGISTER_DISPATCH(asin_stub, &asin_kernel_cuda);
REGISTER_DISPATCH(sin_stub, &sin_kernel_cuda);
REGISTER_DISPATCH(cos_stub, &cos_kernel_cuda);
REGISTER_DISPATCH(sinh_stub, &sinh_kernel_cuda);
REGISTER_DISPATCH(cosh_stub, &cosh_kernel_cuda);
REGISTER_DISPATCH(tanh_stub, &tanh_kernel_cuda);

}} // namespace at::native
